#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void test01()
{
    printf("The block id is %d --- The thread id is %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    // Launch 1 block with 1 thread
    test01<<<1,4>>>();


    return 0;
}