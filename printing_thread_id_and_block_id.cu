#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void kernel()
{
  // print the block ids and thread ids.
  printf("\nThe block id is %d----The thread id is %d\n",blockIdx.x,threadIdx.x);
}
int main()
{
  // PARAMS: (num_of_blk,no_of_thread)
  kernel<<<1,512>>>();

  return 0;
}
