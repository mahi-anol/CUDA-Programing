#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void test01()
{
    int warp_ID_Value=0;
    warp_ID_Value=threadIdx.x/32;

    printf("The block id is %d --- The thread id is %d --The wardID %d\n", blockIdx.x, threadIdx.x,warp_ID_Value);
}

int main()
{
    // Launch 1 block with 1 thread
    test01<<<1,4>>>();


    return 0;
}